#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ising.h"

#define			BLOCK						256
#define			THREAD					32

#define			idx(i,j)				((n+i)%n)*n + (n+j)%n
#define 		Xmat(i,j)				X[ idx(i,j) ]
#define			Ymat(i,j)				Y[ idx(i,j) ]
#define			weight(i,j)			w[ (2+i)*5 + (2+j) ]


__global__ void kernel(int* Y, int *X, double *w, int k, int n) {

	int stride = gridDim.y * blockDim.y,
			i = threadIdx.x + blockIdx.x * blockDim.x,
			j = threadIdx.y + blockIdx.y * blockDim.y;

	
	for(; i < n; i += stride ) {
		for(; j < n; j += stride ) {

			double ws = 0;
			for(int l = -2; l <= 2; l++)
				for(int m = -2; m <= 2; m++)
					ws += weight(l,m) * Xmat(i+l, j+m);

			Ymat(i,j) = update(Xmat(i,j), ws);
		}
	}

}

__host__ void ising(int* G, double* w, int k, int n) {

	int *X, *Y;
	double *d_w;

	hipMalloc(&X, n*n*sizeof(int));
	hipMalloc(&Y, n*n*sizeof(int));
	hipMalloc(&d_w, 5*5*sizeof(double));

	hipMemcpy(d_w, w, 5*5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(Y, G, n*n*sizeof(int), hipMemcpyHostToDevice);

	dim3 N(BLOCK,BLOCK),
			 M(THREAD,THREAD);

	while(k > 0) {
		
		swap_mat( &X, &Y );

		kernel<<<N,M>>>(Y,X,d_w,k,n);

		hipDeviceSynchronize();

		k--;
	}

	hipMemcpy(G, Y, n*n*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(X); hipFree(Y);
	hipFree(d_w);

	return;

}


__device__ int update(int old_value, double ws) {

	int sign = sgn(ws);

	return (old_value*(sign == 0) + sign*(sign!=0));

}

static void swap_mat(int **G, int **H) {

	int *tmp = *H;
	*H = *G;
	*G = tmp;


}

__device__ int sgn (double d){

	const double acc=1.0e-8;

	return (( d > acc) - (d < -acc));
}
