#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ising.h"

#define		MAX_THREADS			32
#define		MAX_BLOCKS			256

#define			idx(i,j)				((n+i)%n)*n + (n+j)%n
#define 		Xmat(i,j)				X[ idx(i,j) ]
#define			Ymat(i,j)				Y[ idx(i,j) ]
#define			weight(i,j)			w[ (2+i)*5 + (2+j) ]


__global__ void kernel( int *X, int* Y, double *w, int n) {

	int row_index = blockIdx.x * blockDim.x + threadIdx.x,
			coll_index = blockIdx.y * blockDim.y + threadIdx.y,
			row_stride = gridDim.x * blockDim.x,
			coll_stride = gridDim.y * blockDim.y;


			for( int i = row_index; i < n; i += row_stride ) {
				for( int j = coll_index; j < n; j += coll_stride) {

					double ws = 0;
					for(int l = -2; l <= 2; l++)
						for(int m = -2; m <= 2; m++)
							ws += weight(l,m) * Xmat(i+l, j+m);

					Ymat(i,j) = update(Xmat(i,j), ws);

				}
			}
}

void ising(int* G, double* w, int k, int n) {

	int *X, *Y;
	double *d_w;

	hipMalloc(&X, n*n);
	hipMalloc(&Y, n*n);
	hipMalloc(&d_w, 5*5);

	hipMemcpy(Y,G, n*n, hipMemcpyHostToDevice);
	hipMemcpy(d_w, w, 5*5, hipMemcpyHostToDevice);

	while( k > 0) {

		swap_mat( &X, &Y );
	
		dim3 N( MAX_BLOCKS, MAX_BLOCKS ),
				 M( MAX_THREADS, MAX_THREADS );
	
		kernel<<<N,M>>>(X,Y,w,n);

		k--;
	}

	hipMemcpy(G, Y, n*n, hipMemcpyDeviceToHost);

	hipFree(X); hipFree(Y), hipFree(d_w);

	return;

}


__device__ static inline int update(int old_value, double ws) {

	int sign = sgn(ws);

	return (old_value*(sign == 0) + sign*(sign!=0));

}

static void swap_mat(int **G, int **H) {

	int *tmp = *H;
	*H = *G;
	*G = tmp;


}

static inline int sgn (double d){

	const double acc=1.0e-8;

	return (( d > acc) - (d < -acc));
}
