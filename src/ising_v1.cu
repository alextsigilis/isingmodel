#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ising.h"

#define			BLOCK						256
#define			THREAD					32

#define			idx(i,j)				((n+i)%n)*n + (n+j)%n
#define 		Xmat(i,j)				X[ idx(i,j) ]
#define			Ymat(i,j)				Y[ idx(i,j) ]
#define			weight(i,j)			w[ (2+i)*5 + (2+j) ]


__global__ void kernel(int* Y, int *X, double *w, int k, int n) {

	int row_index = blockIdx.x * blockDim.x + threadIdx.x,
			coll_index = blockIdx.y * blockDim.y + threadIdx.y,
			row_stride = gridDim.x * blockDim.x,
			coll_stride = gridDim.y * blockDim.y;


	for(int i = row_index; i < n; i += row_stride) {
		for(int j = coll_index; j < n; j += coll_stride) {

			double ws = 0;
			for(int l = -2; l <= 2; l++)
				for(int m = -2; m <= 2; m++)
					ws += weight(l,m) * Xmat(i+l, j+m);

			Ymat(i,j) = update(Xmat(i,j), ws);	

		}
	}	

}

__host__ void ising(int* G, double* w, int k, int n) {

	int *X, *Y;
	double *d_w;

	hipMallocManaged(&X, n*n);
	hipMallocManaged(&Y, n*n);
	hipMallocManaged(&d_w, 5*5);

	for(int i = 0; i < 5*5; i++) d_w[i] = w[i];
	for(int i = 0; i < n*n; i++) Y[i] = G[i];

	dim3 N(BLOCK,BLOCK),
			 M(THREAD,THREAD);


	while(k > 0) {
		
		swap_mat( &X, &Y );

		kernel<<<N,M>>>(Y,X,d_w,k,n);

		hipDeviceSynchronize();

		k--;
	}

	for(int i = 0; i < n*n; i++) G[i] = Y[i];

	hipFree(X); hipFree(Y);
	hipFree(d_w);

	return;

}


__device__ int update(int old_value, double ws) {

	int sign = sgn(ws);

	return (old_value*(sign == 0) + sign*(sign!=0));

}

static void swap_mat(int **G, int **H) {

	int *tmp = *H;
	*H = *G;
	*G = tmp;


}

__device__ int sgn (double d){

	const double acc=1.0e-8;

	return (( d > acc) - (d < -acc));
}
