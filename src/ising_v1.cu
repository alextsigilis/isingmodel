#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ising.h"

#define		MAX_THREADS			32
#define		MAX_BLOCKS			256

#define			idx(i,j)				((n+i)%n)*n + (n+j)%n
#define 		Xmat(i,j)				X[ idx(i,j) ]
#define			Ymat(i,j)				Y[ idx(i,j) ]
#define			weight(i,j)			w[ (2+i)*5 + (2+j) ]


__global__ void kernel( int *X, int* Y, double *w, int k, int n) {

	int row_index = blockIdx.x * blockDim.x + threadIdx.x,
			coll_index = blockIdx.y * blockDim.y + threadIdx.y,
			row_stride = gridDim.x * blockDim.x,
			coll_stride = gridDim.y * blockDim.y;

	while( k > 0) {

		if(row_index == 0 && coll_index == 0) {
			int *tmp = X;
			X = Y;
			Y = tmp;
		}

		for( int i = row_index; i < n; i += row_stride ) {
			for( int j = coll_index; j < n; j += coll_stride) {

				double ws = 0;
				for(int l = -2; l <= 2; l++)
					for(int m = -2; m <= 2; m++)
						ws += weight(l,m) * Xmat(i+l, j+m);

				// Update Y	
				const double acc=1.0e-8;
				int sgn =  (( ws > acc) - (ws < -acc));
				Ymat(i,j) = (Xmat(i,j)*(sgn == 0) + sgn*(sgn!=0));
			}
		}
	
		k--;
		__syncthreads();
	}

}

void ising(int* G, double* w, int k, int n) {

	int *X, *Y;
	double *d_w;

	hipMalloc(&X, n*n);
	hipMalloc(&Y, n*n);
	hipMalloc(&d_w, 5*5);

	hipMemcpy(Y,G, n*n, hipMemcpyHostToDevice);
	hipMemcpy(d_w, w, 5*5, hipMemcpyHostToDevice);


	
	dim3 N( MAX_BLOCKS, MAX_BLOCKS ),
			 M( MAX_THREADS, MAX_THREADS );
	
	kernel<<<N,M>>>(X,Y,w,k,n);

	
	hipMemcpy(G, Y, n*n, hipMemcpyDeviceToHost);

	hipFree(X); hipFree(Y), hipFree(d_w);

	return;

}

