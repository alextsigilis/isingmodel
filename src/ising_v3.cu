#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "ising.h"

#define	 		GRID_SZ					256
#define			THREADS					32

#define			idx(i,j)				((n+i)%n)*n + (n+j)%n
#define 		Xmat(i,j)				X[ idx(i,j) ]
#define			Ymat(i,j)				Y[ idx(i,j) ]
#define			weight(i,j)			w[ (2+i)*5 + (2+j) ]


__global__ void kernel(int* Y, int *X, double *w, int n) {

	int stride = gridDim.y * blockDim.y,
			i = threadIdx.y + blockIdx.y * blockDim.y,
			j = threadIdx.x + blockIdx.x * blockDim.x;

	
	__shared__ cache[blockDim.y+2][blockDim.x+2];

	
	for(; i < n; i += stride ) {
		for(; j < n; j += stride ) {

		

		// Load threads element
		cache[threadIdx.x+1][threadIdx.y+1] = Xmat(i,j);

		// Load the surrounding elements
			
			double ws = 0;
			for(int l = -2; l <= 2; l++)
				for(int m = -2; m <= 2; m++)
					ws += weight(l,m) * Xmat(i+l, j+m);

			Ymat(i,j) = update(Xmat(i,j), ws);
		}
	}

}

__host__ void ising(int* G, double* w, int k, int n) {

	int *X, *Y;
	double *d_w;

	hipMalloc(&X, n*n*sizeof(int));
	hipMalloc(&Y, n*n*sizeof(int));
	hipMalloc(&d_w, 5*5*sizeof(double));

	hipMemcpy(d_w, w, 5*5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(Y, G, n*n*sizeof(int), hipMemcpyHostToDevice);

	dim3 N( GRID_SZ, GRID_SZ ),
			 M( THREADS, THREADS );

	while(k > 0) {
		
		swap_mat( &X, &Y );

		kernel<<<N,M>>>(Y,X,d_w,n);

		hipDeviceSynchronize();

		k--;
	}

	hipMemcpy(G, Y, n*n*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(X); hipFree(Y);
	hipFree(d_w);

	return;

}


__device__ static inline int update(int old_value, double ws) {

	int sign = sgn(ws);

	return (old_value*(sign == 0) + sign*(sign!=0));

}

static inline void swap_mat(int **G, int **H) {

	int *tmp = *H;
	*H = *G;
	*G = tmp;


}

__device__ inline int sgn (double d){

	const double acc=1.0e-8;

	return (( d > acc) - (d < -acc));
}
